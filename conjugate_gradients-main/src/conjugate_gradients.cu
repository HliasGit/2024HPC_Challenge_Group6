#include <cstdio>
#include <cstdlib>
#include <cmath>
// #include "cuda_kernels.cuh"
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define BLOCK_DIM_VET 32

bool read_matrix_from_file(const char * filename, double ** matrix_out, size_t * num_rows_out, size_t * num_cols_out)
{
    double * matrix;
    size_t num_rows;
    size_t num_cols;

    FILE * file = fopen(filename, "rb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fread(&num_rows, sizeof(size_t), 1, file);
    fread(&num_cols, sizeof(size_t), 1, file);
    matrix = new double[num_rows * num_cols];
    fread(matrix, sizeof(double), num_rows * num_cols, file);

    *matrix_out = matrix;
    *num_rows_out = num_rows;
    *num_cols_out = num_cols;

    fclose(file);

    return true;
}



bool write_matrix_to_file(const char * filename, const double * matrix, size_t num_rows, size_t num_cols)
{
    FILE * file = fopen(filename, "wb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fwrite(&num_rows, sizeof(size_t), 1, file);
    fwrite(&num_cols, sizeof(size_t), 1, file);
    fwrite(matrix, sizeof(double), num_rows * num_cols, file);

    fclose(file);

    return true;
}



void print_matrix(const double * matrix, size_t num_rows, size_t num_cols, FILE * file = stdout)
{
    fprintf(file, "%zu %zu\n", num_rows, num_cols);
    for(size_t r = 0; r < num_rows; r++)
    {
        for(size_t c = 0; c < num_cols; c++)
        {
            double val = matrix[r * num_cols + c];
            printf("%+6.3f ", val);
        }
        printf("\n");
    }
}



double dot(const double * x, const double * y, size_t size, hipblasHandle_t handle)
{
    // Scalar product
    double result;
    hipblasDdot(handle, size, x, size, y, size, &result);
    return result;
}



void axpby(double * alpha, const double * x, double * beta, double * y, size_t size, hipblasHandle_t handle)
{
    // y = alpha * x + beta * y
    hipblasDaxpy(handle, size, alpha, x, size, y, size);
}



void gemv(double * alpha, const double * A, const double * x, double * beta, double * y, size_t size, hipblasHandle_t handle)
{
    // y = alpha * A * x + beta * y;
    hipblasDgemv(handle, HIPBLAS_OP_N, size, size, alpha, A, size, x, size, beta, y, size);
}

__global__ void init_cg_solver(double * x, double * p, const double * r, size_t size)
{   
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) 
    {
		x[index] = 0.0;
        p[index] = r[index];
    }
}
__global__ void init_alpha_beta(double * a, double * b)
{
    *a = 1.0;
    *b = 0.0;
}
__global__ void copy_value(double * a, const double * b)
{
    *a = *b;
}   
__global__ void a_frac_b(const double * x, const double * y, double * z)
{
    *z = *x / *y;
}
__global__ void inv_alpha(const double * alpha, double * alpha_inv)
{
    *alpha_inv = *alpha;
}
void conjugate_gradients(const double * d_A, double * d_x, double * d_p, double * d_r,  size_t size, int max_iters, double rel_error)
{
    // Create paramters
    double* d_beta;
    double* d_alpha; 
    double* d_alpha_;
    double* a;
    double* b; 
	double* d_rr_new;
	double* d_rr;
	double* d_bb;
    double* d_Ap;
    

    double * tmp;
    // Relative residual
    double h_bb;
    double h_rr;

    int num_iters;
    hipMalloc((void **) &d_Ap, size * sizeof(double));
	hipMalloc((void **) &d_beta, sizeof(double));
    hipMalloc((void **) &tmp, sizeof(double));
    hipMalloc((void **) &a, sizeof(double));
    hipMalloc((void **) &b, sizeof(double));
	hipMalloc((void **) &d_alpha, sizeof(double));
    hipMalloc((void **) &d_alpha_, sizeof(double));
	hipMalloc((void **) &d_rr_new, sizeof(double));
	hipMalloc((void **) &d_rr, sizeof(double));
	hipMalloc((void **) &d_bb, sizeof(double));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    dim3 vec_block_dim(BLOCK_DIM_VET);
	dim3 vec_grid_dim((size + BLOCK_DIM_VET - 1) / BLOCK_DIM_VET);
    // Set alpha = 1 and beta = 0
    init_alpha_beta<<<1, 1>>>(a, b);
    // Init solver with x0 = 0 and p = r
    init_cg_solver <<<vec_grid_dim, vec_block_dim>>> (d_x, d_p, d_r, size);
    // Coompute r * r = b*b
    hipblasDdot(handle, size, d_r, size, d_r, size, d_rr);
    copy_value <<<1,1>>> (d_bb, d_rr);

    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        // alpha(k) = rr / p A p
        hipblasDgemv(handle, HIPBLAS_OP_N, size, size, a, d_A, size, d_p, size, b, d_Ap, size);
        hipblasDdot(handle, size, d_p, size, d_Ap, size, tmp);
        a_frac_b<<<1, 1>>> (d_rr, tmp, d_alpha);
        
        inv_alpha<<<1,1>>> (d_alpha, d_alpha_);

        // x_(k+1) = x(k) + alpha * p
        hipblasDaxpy(handle, size, d_alpha, d_p, size, d_x, size);
        //r(k+1) = r(k) - alpha * A * p
        hipblasDaxpy(handle, size, d_alpha_, d_Ap, size, d_r, size);
        // beta(k) = r(k+1)r(k+1) / r(k)r(k)
        hipblasDdot(handle, size, d_r, size, d_r, size, d_rr_new);
        a_frac_b <<<1, 1>>> (d_rr_new, d_rr, d_beta);

        // Update d_rr
        copy_value <<<1, 1>>> (d_rr, d_rr_new);

        // Compute relative residual
        a_frac_b <<<1, 1>>> (d_rr, d_bb, tmp);
        hipMemcpy(d_rr, &h_rr, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(d_bb, &h_bb, sizeof(double), hipMemcpyDeviceToHost);
        // Stopping criteria
        if(std::sqrt(h_rr / h_bb) < rel_error) 
            break; 
        // p(k+1) = r(k+1) + beta * p
        hipblasDaxpy(handle, size, d_beta, d_p, size, d_r, size);
    }

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(h_rr / h_bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(h_rr / h_bb));
    }
}

int main(int argc, char ** argv)
{
    printf("Usage: ./random_matrix input_file_matrix.bin input_file_rhs.bin output_file_sol.bin max_iters rel_error\n");
    printf("All parameters are optional and have default values\n");
    printf("\n");

    const char * input_file_matrix = "io/matrix.bin";
    const char * input_file_rhs = "io/rhs.bin";
    const char * output_file_sol = "io/sol.bin";
    int max_iters = 1000;
    double rel_error = 1e-9;

    if(argc > 1) input_file_matrix = argv[1];
    if(argc > 2) input_file_rhs = argv[2];
    if(argc > 3) output_file_sol = argv[3];
    if(argc > 4) max_iters = atoi(argv[4]);
    if(argc > 5) rel_error = atof(argv[5]);

    printf("Command line arguments:\n");
    printf("  input_file_matrix: %s\n", input_file_matrix);
    printf("  input_file_rhs:    %s\n", input_file_rhs);
    printf("  output_file_sol:   %s\n", output_file_sol);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("\n");



    double * matrix;
    double * rhs;
    size_t size;

    {
        printf("Reading matrix from file ...\n");
        size_t matrix_rows;
        size_t matrix_cols;
        bool success_read_matrix = read_matrix_from_file(input_file_matrix, &matrix, &matrix_rows, &matrix_cols);
        if(!success_read_matrix)
        {
            fprintf(stderr, "Failed to read matrix\n");
            return 1;
        }
        printf("Done\n");
        printf("\n");

        printf("Reading right hand side from file ...\n");
        size_t rhs_rows;
        size_t rhs_cols;
        bool success_read_rhs = read_matrix_from_file(input_file_rhs, &rhs, &rhs_rows, &rhs_cols);
        if(!success_read_rhs)
        {
            fprintf(stderr, "Failed to read right hand side\n");
            return 2;
        }
        printf("Done\n");
        printf("\n");

        if(matrix_rows != matrix_cols)
        {
            fprintf(stderr, "Matrix has to be square\n");
            return 3;
        }
        if(rhs_rows != matrix_rows)
        {
            fprintf(stderr, "Size of right hand side does not match the matrix\n");
            return 4;
        }
        if(rhs_cols != 1)
        {
            fprintf(stderr, "Right hand side has to have just a single column\n");
            return 5;
        }

        size = matrix_rows;
    }

    printf("Solving the system ...\n");
    double * sol = new double[size];

    // allocate device memory
	double* d_A;
	double* d_b;
	double* d_x;
	double* d_p;
	double* d_r;
	double* d_temp;
	hipMalloc((void **) &d_A, size * size * sizeof(double));
	hipMalloc((void **) &d_b, size * sizeof(double));
	hipMalloc((void **) &d_x, size * sizeof(double));
	hipMalloc((void **) &d_p, size * sizeof(double));
	hipMalloc((void **) &d_r, size * sizeof(double));
	hipMalloc((void **) &d_temp, size * sizeof(double));

    // copy host memory to device
	hipMemcpy(d_A, matrix, size * size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, sol, size * sizeof(double), hipMemcpyHostToDevice);
	// assume x0 = 0
	hipMemcpy(d_p, rhs, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, size * sizeof(double), hipMemcpyHostToDevice);

    conjugate_gradients(d_A, d_x, d_p, d_r, size, max_iters, rel_error);
    printf("Done\n");
    printf("\n");

    printf("Writing solution to file ...\n");
    bool success_write_sol = write_matrix_to_file(output_file_sol, sol, size, 1);
    if(!success_write_sol)
    {
        fprintf(stderr, "Failed to save solution\n");
        return 6;
    }
    printf("Done\n");
    printf("\n");

    delete[] matrix;
    delete[] rhs;
    delete[] sol;

    // Clean device memory
    // cleanup memory device
	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_p);
	hipFree(d_r);

    printf("Finished successfully\n");

    return 0;
}