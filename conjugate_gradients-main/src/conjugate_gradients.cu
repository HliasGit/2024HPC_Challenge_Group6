#include "hip/hip_runtime.h"

#include "utilities.cuh"

#define BLOCK_DIM_VET 32

void conjugate_gradients(const double * d_A, double * d_x, double * d_p, double * d_r,  size_t size, int max_iters, double rel_error)
{
    double* d_beta;
    double* d_alpha; 
    double* d_alpha_;
    double* a;
    double* b; 
	double* d_rr_new;
	double* d_rr;
	double* d_bb;
    double* d_Ap;
    double * tmp;

    // Host relative residual
    double h_bb;
    double h_rr;

    // Allocate device memory 
    hipMalloc((void **) &d_Ap, size * sizeof(double));
	hipMalloc((void **) &d_beta, sizeof(double));
    hipMalloc((void **) &tmp, sizeof(double));
    hipMalloc((void **) &a, sizeof(double));
    hipMalloc((void **) &b, sizeof(double));
	hipMalloc((void **) &d_alpha, sizeof(double));
    hipMalloc((void **) &d_alpha_, sizeof(double));
	hipMalloc((void **) &d_rr_new, sizeof(double));
	hipMalloc((void **) &d_rr, sizeof(double));
	hipMalloc((void **) &d_bb, sizeof(double));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) 
    {
        printf ("CUBLAS initialization failed\n");
        return;
    }
    // Set only GPU pointers for cuBLAS functions
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    // Choose block and grid dimensions
    dim3 vec_block_dim(BLOCK_DIM_VET);
	dim3 vec_grid_dim((size + BLOCK_DIM_VET - 1) / BLOCK_DIM_VET);
    // Set alpha = 1 and beta = 0
    init_alpha_beta<<<1, 1>>>(a, b);
    // Init solver with x0 = 0 and p = r
    init_cg_solver <<<vec_grid_dim, vec_block_dim>>> (d_x, d_p, d_r, size);
    
    // Coompute r * r = b*b
    hipblasDdot(handle, size, d_r, 1, d_r, 1, d_rr);
    copy_value <<<1,1>>> (d_bb, d_rr);
    hipMemcpy(&h_bb, d_bb, sizeof(double), hipMemcpyDeviceToHost);

    int num_iters;
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        // alpha(k) = rr / p A p
        hipblasDgemv(handle, HIPBLAS_OP_N, size, size, a, d_A, size, d_p, 1, b, d_Ap, 1);
        hipblasDdot(handle, size, d_p, 1, d_Ap, 1, tmp);
        a_frac_b<<<1, 1>>> (d_rr, tmp, d_alpha);
        // Compute -alpha
        inv_alpha<<<1, 1>>> (d_alpha, d_alpha_);

        // x(k+1) = x(k) + alpha * p
        hipblasDaxpy(handle, size, d_alpha, d_p, 1, d_x, 1);

        //r(k+1) = r(k) - alpha * A * p
        hipblasDaxpy(handle, size, d_alpha_, d_Ap, 1, d_r, 1);

        //beta(k) = r(k+1)r(k+1) / r(k)r(k)
        hipblasDdot(handle, size, d_r, 1, d_r, 1, d_rr_new);
        a_frac_b <<<1, 1>>> (d_rr_new, d_rr, d_beta);

        // Update d_rr
        copy_value <<<1, 1>>> (d_rr, d_rr_new);

        // Synchronize host's relative residuals
        hipMemcpy(&h_rr, d_rr, sizeof(double), hipMemcpyDeviceToHost);
        // Stopping criteria
        if(std::sqrt(h_rr / h_bb) < rel_error) 
            break; 
        // p(k+1) = r(k+1) + beta * p(k)
        scalar_vet <<<vec_grid_dim, vec_block_dim>>> (d_beta, d_p, size);
        hipblasDaxpy(handle, size, a, d_r, 1, d_p, 1);
    }

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(h_rr / h_bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(h_rr / h_bb));
    }

    
}

int main(int argc, char ** argv)
{
    printf("Usage: ./random_matrix input_file_matrix.bin input_file_rhs.bin output_file_sol.bin max_iters rel_error\n");
    printf("All parameters are optional and have default values\n");
    printf("\n");

    const char * input_file_matrix = "io/matrix.bin";
    const char * input_file_rhs = "io/rhs.bin";
    const char * output_file_sol = "io/sol.bin";
    int max_iters = 1000;
    double rel_error = 1e-9;

    if(argc > 1) input_file_matrix = argv[1];
    if(argc > 2) input_file_rhs = argv[2];
    if(argc > 3) output_file_sol = argv[3];
    if(argc > 4) max_iters = atoi(argv[4]);
    if(argc > 5) rel_error = atof(argv[5]);

    printf("Command line arguments:\n");
    printf("  input_file_matrix: %s\n", input_file_matrix);
    printf("  input_file_rhs:    %s\n", input_file_rhs);
    printf("  output_file_sol:   %s\n", output_file_sol);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("\n");



    double * matrix;
    double * rhs;
    size_t size;

    {
        printf("Reading matrix from file ...\n");
        size_t matrix_rows;
        size_t matrix_cols;
        bool success_read_matrix = read_matrix_from_file(input_file_matrix, &matrix, &matrix_rows, &matrix_cols);
        if(!success_read_matrix)
        {
            fprintf(stderr, "Failed to read matrix\n");
            return 1;
        }
        printf("Done\n");
        printf("\n");

        printf("Reading right hand side from file ...\n");
        size_t rhs_rows;
        size_t rhs_cols;
        bool success_read_rhs = read_matrix_from_file(input_file_rhs, &rhs, &rhs_rows, &rhs_cols);
        if(!success_read_rhs)
        {
            fprintf(stderr, "Failed to read right hand side\n");
            return 2;
        }
        printf("Done\n");
        printf("\n");

        if(matrix_rows != matrix_cols)
        {
            fprintf(stderr, "Matrix has to be square\n");
            return 3;
        }
        if(rhs_rows != matrix_rows)
        {
            fprintf(stderr, "Size of right hand side does not match the matrix\n");
            return 4;
        }
        if(rhs_cols != 1)
        {
            fprintf(stderr, "Right hand side has to have just a single column\n");
            return 5;
        }

        size = matrix_rows;
    }

    printf("Solving the system ...\n");
    double * sol = new double[size];

    // Allocate device memory
	double* d_A;
	double* d_b;
	double* d_x;
	double* d_p;
	double* d_r;
	double* d_temp;
	hipMalloc((void **) &d_A, size * size * sizeof(double));
	hipMalloc((void **) &d_b, size * sizeof(double));
	hipMalloc((void **) &d_x, size * sizeof(double));
	hipMalloc((void **) &d_p, size * sizeof(double));
	hipMalloc((void **) &d_r, size * sizeof(double));
	hipMalloc((void **) &d_temp, size * sizeof(double));

    // Copy host memory to device
	hipMemcpy(d_A, matrix, size * size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, sol, size * sizeof(double), hipMemcpyHostToDevice);
	// Assume x0 = 0
	hipMemcpy(d_p, rhs, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, size * sizeof(double), hipMemcpyHostToDevice);
    // Solve Ax = b 
    conjugate_gradients(d_A, d_x, d_p, d_r, size, max_iters, rel_error);

    printf("Done\n");
    printf("\n");

    printf("Writing solution to file ...\n");
    bool success_write_sol = write_matrix_to_file(output_file_sol, sol, size, 1);
    if(!success_write_sol)
    {
        fprintf(stderr, "Failed to save solution\n");
        return 6;
    }
    printf("Done\n");
    printf("\n");

    delete[] matrix;
    delete[] rhs;
    delete[] sol;

    // Clean device memory
    // cleanup memory device
	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_p);
	hipFree(d_r);

    printf("Finished successfully\n");

    return 0;
}