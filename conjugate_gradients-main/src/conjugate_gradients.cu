#include <cstdio>
#include <cstdlib>
#include <cmath>
// #include "cuda_kernels.cuh"
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define BLOCK_DIM_VET 32

bool read_matrix_from_file(const char * filename, double ** matrix_out, size_t * num_rows_out, size_t * num_cols_out)
{
    double * matrix;
    size_t num_rows;
    size_t num_cols;

    FILE * file = fopen(filename, "rb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fread(&num_rows, sizeof(size_t), 1, file);
    fread(&num_cols, sizeof(size_t), 1, file);
    matrix = new double[num_rows * num_cols];
    fread(matrix, sizeof(double), num_rows * num_cols, file);

    *matrix_out = matrix;
    *num_rows_out = num_rows;
    *num_cols_out = num_cols;

    fclose(file);

    return true;
}



bool write_matrix_to_file(const char * filename, const double * matrix, size_t num_rows, size_t num_cols)
{
    FILE * file = fopen(filename, "wb");
    if(file == nullptr)
    {
        fprintf(stderr, "Cannot open output file\n");
        return false;
    }

    fwrite(&num_rows, sizeof(size_t), 1, file);
    fwrite(&num_cols, sizeof(size_t), 1, file);
    fwrite(matrix, sizeof(double), num_rows * num_cols, file);

    fclose(file);

    return true;
}



void print_matrix(const double * matrix, size_t num_rows, size_t num_cols, FILE * file = stdout)
{
    fprintf(file, "%zu %zu\n", num_rows, num_cols);
    for(size_t r = 0; r < num_rows; r++)
    {
        for(size_t c = 0; c < num_cols; c++)
        {
            double val = matrix[r * num_cols + c];
            printf("%+6.3f ", val);
        }
        printf("\n");
    }
}
void print_vet(const double * vet, size_t num_rows, FILE * file = stdout)
{
    fprintf(file, "%z\t", num_rows);
    for(size_t r = 0; r < num_rows; r++)
    {
        double val = vet[r];
        printf("%+6.3f\t ", val);
    }
    printf("\n");
}


__global__ void init_cg_solver(double * x, double * p, const double * r, size_t size)
{   
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) 
    {
		x[index] = 0.0;
        p[index] = r[index];
    }
}
__global__ void init_alpha_beta(double * a, double * b)
{
    *a = 1.0;
    *b = 0.0;
}
__global__ void copy_value(double * a, const double * b)
{
    *a = *b;
}   
__global__ void a_frac_b(const double * x, const double * y, double * z)
{
    *z = (*x) / (*y);
}
__global__ void inv_alpha(const double * alpha, double * alpha_inv)
{
    *alpha_inv = - (*alpha);
}

__global__ void scalar_vet(const double * alpha, double * x, size_t size)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) 
    {
		x[index] = (*alpha) * x[index];
    }
}

void print_matrix_mcpy(const double * d_A, size_t rows, size_t columns)
{   

    if (columns > 0)
    {
        double *h_A = (double *) malloc(rows * columns * sizeof(double));
        hipMemcpy(h_A, d_A, rows*columns, hipMemcpyDeviceToHost);
        print_matrix(h_A, rows, columns);
    }
    else
    {
        double *h_A = (double *) malloc(rows * sizeof(double));
        hipError_t err = hipMemcpy(h_A, d_A, rows * sizeof(double), hipMemcpyDeviceToHost);
        print_vet(h_A, rows);
    }
}
void print_scalar(const double * scalar)
{
        double *h_A = (double *) malloc(sizeof(double));
        hipError_t err = hipMemcpy(h_A, scalar, sizeof(double), hipMemcpyDeviceToHost);
        printf("%+6.5f\n", *h_A);
}
void conjugate_gradients(const double * d_A, double * d_x, double * d_p, double * d_r,  size_t size, int max_iters, double rel_error)
{
    double* d_beta;
    double* d_alpha; 
    double* d_alpha_;
    double* a;
    double* b; 
	double* d_rr_new;
	double* d_rr;
	double* d_bb;
    double* d_Ap;
    double * tmp;

    // Host relative residual
    double h_bb;
    double h_rr;

    // Allocate device memory 
    hipMalloc((void **) &d_Ap, size * sizeof(double));
	hipMalloc((void **) &d_beta, sizeof(double));
    hipMalloc((void **) &tmp, sizeof(double));
    hipMalloc((void **) &a, sizeof(double));
    hipMalloc((void **) &b, sizeof(double));
	hipMalloc((void **) &d_alpha, sizeof(double));
    hipMalloc((void **) &d_alpha_, sizeof(double));
	hipMalloc((void **) &d_rr_new, sizeof(double));
	hipMalloc((void **) &d_rr, sizeof(double));
	hipMalloc((void **) &d_bb, sizeof(double));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) 
    {
        printf ("CUBLAS initialization failed\n");
        return;
    }
    // Set only GPU pointers for cuBLAS functions
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    // Choose block and grid dimensions
    dim3 vec_block_dim(BLOCK_DIM_VET);
	dim3 vec_grid_dim((size + BLOCK_DIM_VET - 1) / BLOCK_DIM_VET);
    // Set alpha = 1 and beta = 0
    init_alpha_beta<<<1, 1>>>(a, b);
    // Init solver with x0 = 0 and p = r
    init_cg_solver <<<vec_grid_dim, vec_block_dim>>> (d_x, d_p, d_r, size);
    
    // Coompute r * r = b*b
    hipblasDdot(handle, size, d_r, 1, d_r, 1, d_rr);
    copy_value <<<1,1>>> (d_bb, d_rr);
    hipMemcpy(&h_bb, d_bb, sizeof(double), hipMemcpyDeviceToHost);

    int num_iters;
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        // alpha(k) = rr / p A p
        hipblasDgemv(handle, HIPBLAS_OP_N, size, size, a, d_A, size, d_p, 1, b, d_Ap, 1);
        hipblasDdot(handle, size, d_p, 1, d_Ap, 1, tmp);
        a_frac_b<<<1, 1>>> (d_rr, tmp, d_alpha);
        // Compute -alpha
        inv_alpha<<<1, 1>>> (d_alpha, d_alpha_);

        // x(k+1) = x(k) + alpha * p
        hipblasDaxpy(handle, size, d_alpha, d_p, 1, d_x, 1);

        //r(k+1) = r(k) - alpha * A * p
        hipblasDaxpy(handle, size, d_alpha_, d_Ap, 1, d_r, 1);

        //beta(k) = r(k+1)r(k+1) / r(k)r(k)
        hipblasDdot(handle, size, d_r, 1, d_r, 1, d_rr_new);
        a_frac_b <<<1, 1>>> (d_rr_new, d_rr, d_beta);

        // Update d_rr
        copy_value <<<1, 1>>> (d_rr, d_rr_new);

        // Synchronize host's relative residuals
        hipMemcpy(&h_rr, d_rr, sizeof(double), hipMemcpyDeviceToHost);
        // Stopping criteria
        if(std::sqrt(h_rr / h_bb) < rel_error) 
            break; 
        // p(k+1) = r(k+1) + beta * p(k)
        scalar_vet <<<vec_grid_dim, vec_block_dim>>> (d_beta, d_p, size);
        hipblasDaxpy(handle, size, a, d_r, 1, d_p, 1);
    }

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(h_rr / h_bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(h_rr / h_bb));
    }

    
}

int main(int argc, char ** argv)
{
    printf("Usage: ./random_matrix input_file_matrix.bin input_file_rhs.bin output_file_sol.bin max_iters rel_error\n");
    printf("All parameters are optional and have default values\n");
    printf("\n");

    const char * input_file_matrix = "io/matrix.bin";
    const char * input_file_rhs = "io/rhs.bin";
    const char * output_file_sol = "io/sol.bin";
    int max_iters = 1000;
    double rel_error = 1e-9;

    if(argc > 1) input_file_matrix = argv[1];
    if(argc > 2) input_file_rhs = argv[2];
    if(argc > 3) output_file_sol = argv[3];
    if(argc > 4) max_iters = atoi(argv[4]);
    if(argc > 5) rel_error = atof(argv[5]);

    printf("Command line arguments:\n");
    printf("  input_file_matrix: %s\n", input_file_matrix);
    printf("  input_file_rhs:    %s\n", input_file_rhs);
    printf("  output_file_sol:   %s\n", output_file_sol);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("\n");



    double * matrix;
    double * rhs;
    size_t size;

    {
        printf("Reading matrix from file ...\n");
        size_t matrix_rows;
        size_t matrix_cols;
        bool success_read_matrix = read_matrix_from_file(input_file_matrix, &matrix, &matrix_rows, &matrix_cols);
        if(!success_read_matrix)
        {
            fprintf(stderr, "Failed to read matrix\n");
            return 1;
        }
        printf("Done\n");
        printf("\n");

        printf("Reading right hand side from file ...\n");
        size_t rhs_rows;
        size_t rhs_cols;
        bool success_read_rhs = read_matrix_from_file(input_file_rhs, &rhs, &rhs_rows, &rhs_cols);
        if(!success_read_rhs)
        {
            fprintf(stderr, "Failed to read right hand side\n");
            return 2;
        }
        printf("Done\n");
        printf("\n");

        if(matrix_rows != matrix_cols)
        {
            fprintf(stderr, "Matrix has to be square\n");
            return 3;
        }
        if(rhs_rows != matrix_rows)
        {
            fprintf(stderr, "Size of right hand side does not match the matrix\n");
            return 4;
        }
        if(rhs_cols != 1)
        {
            fprintf(stderr, "Right hand side has to have just a single column\n");
            return 5;
        }

        size = matrix_rows;
    }

    printf("Solving the system ...\n");
    double * sol = new double[size];

    // Allocate device memory
	double* d_A;
	double* d_b;
	double* d_x;
	double* d_p;
	double* d_r;
	double* d_temp;
	hipMalloc((void **) &d_A, size * size * sizeof(double));
	hipMalloc((void **) &d_b, size * sizeof(double));
	hipMalloc((void **) &d_x, size * sizeof(double));
	hipMalloc((void **) &d_p, size * sizeof(double));
	hipMalloc((void **) &d_r, size * sizeof(double));
	hipMalloc((void **) &d_temp, size * sizeof(double));

    // Copy host memory to device
	hipMemcpy(d_A, matrix, size * size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, sol, size * sizeof(double), hipMemcpyHostToDevice);
	// Assume x0 = 0
	hipMemcpy(d_p, rhs, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, size * sizeof(double), hipMemcpyHostToDevice);
    // Solve Ax = b 
    conjugate_gradients(d_A, d_x, d_p, d_r, size, max_iters, rel_error);

    printf("Done\n");
    printf("\n");

    printf("Writing solution to file ...\n");
    bool success_write_sol = write_matrix_to_file(output_file_sol, sol, size, 1);
    if(!success_write_sol)
    {
        fprintf(stderr, "Failed to save solution\n");
        return 6;
    }
    printf("Done\n");
    printf("\n");

    delete[] matrix;
    delete[] rhs;
    delete[] sol;

    // Clean device memory
    // cleanup memory device
	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_p);
	hipFree(d_r);

    printf("Finished successfully\n");

    return 0;
}