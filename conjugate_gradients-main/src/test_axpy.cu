#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

int main(void)
{
    const int nvals = 2;
    const size_t sz = sizeof(double) * (size_t)nvals;
    double x[nvals], y[nvals];
    double *x_, *y_, *result_;
    double result=0., resulth=0.;

    // for(int i=0; i<nvals; i++) {
    //     x[i] = y[i] = 1.0;
    //     resulth += x[i] * y[i];
    // }
    x[0] = -0.569;
    x[1] = +0.680;

    y[0] = +0.014;
    y[1] = 0.012;

    double * alpha_, * beta_;
    double alpha = 1, beta = 1;

    hipblasHandle_t h;
    hipblasStatus_t stat;
    hipblasCreate(&h);
    hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);
    
    hipMalloc( (void **)(&x_), sz);
    hipMalloc( (void **)(&y_), sz);

    hipMalloc( (void **)(&alpha_), sizeof(double));


    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);

    hipMemcpy(alpha_, &alpha, sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(y, y_, sz, hipMemcpyDeviceToHost);
    for (int i = 0; i < nvals; i++)
        printf("%f\n", y[i]);
    printf("\n");

    stat = hipblasDaxpy(h, nvals, alpha_, x_, 1, y_, 1);
    {    
        if (stat == HIPBLAS_STATUS_SUCCESS)
        {
            printf("La chiamata a hipblasDgemv è stata eseguita con successo.\n");
        }
        else if (stat == HIPBLAS_STATUS_NOT_INITIALIZED)
        {
            printf("Errore: cuBLAS non è stato inizializzato correttamente.\n");
        }
        else if (stat == HIPBLAS_STATUS_INVALID_VALUE)
        {
            printf("Errore: uno o più parametri di input sono invalidi.\n");
        }
        else if (stat == HIPBLAS_STATUS_ALLOC_FAILED)
        {
            printf("Errore: l'allocazione di memoria sulla GPU ha fallito.\n");
        }
    }
    hipMemcpy(y, y_, sz, hipMemcpyDeviceToHost);

    for (int i = 0; i < nvals; i++)
        printf("%f\n", y[i]);
    printf("\n");

    hipMemcpy(x, x_, sz, hipMemcpyDeviceToHost);

    hipblasDestroy(h);
    hipFree(alpha_);

    return 0;
}