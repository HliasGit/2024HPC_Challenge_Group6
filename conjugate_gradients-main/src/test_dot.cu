#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

int main(void)
{
    const int nvals = 2;
    const size_t sz = sizeof(double) * (size_t)nvals;
    double x[nvals], y[nvals];
    double *x_, *y_, *result_;
    double result=0., resulth=0.;

    for(int i=0; i<nvals; i++) {
        x[i] = y[i] = 2.0;
        resulth += x[i] * y[i];
    }

    hipblasHandle_t h;
    hipblasCreate(&h);
    hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);
    
    hipMalloc( (void **)(&x_), sz);
    hipMalloc( (void **)(&y_), sz);
    hipMalloc( (void **)(&result_), sizeof(double) );

    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);

    hipblasDdot(h, nvals, x_, 1, x_, 1, result_);

    hipMemcpy(&result, result_, sizeof(double), hipMemcpyDeviceToHost);

    printf("%f %f\n", resulth, result);

    hipblasDestroy(h);
    return 0;
}