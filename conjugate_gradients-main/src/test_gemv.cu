#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

int main(void)
{
    const int nvals = 2;
    const size_t sz = sizeof(double) * (size_t)nvals;
    double x[nvals], y[nvals];
    double A[nvals * nvals];

    double alpha = 1, beta = 1;
    double *alpha_, *beta_;

    double *A_;
    double *x_, *y_;

    for (int i = 0; i < nvals; i++)
    {
        x[i] = y[i] = 1.0;

        for (int j = 0; j < nvals; j++)
        {
            A[i * nvals + j] = 1.0;
        }
    }

    hipblasHandle_t h;
    hipblasCreate(&h);
    hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);

    hipMalloc((void **)(&x_), sz);
    hipMalloc((void **)(&y_), sz);
    hipMalloc((void **)(&A_), sz * nvals);
    hipMalloc((void **)(&alpha_), sizeof(double));
    hipMalloc((void **)(&beta_), sizeof(double));

    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);
    hipMemcpy(alpha_, &alpha, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(beta_, &beta, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(A_, A, sz * nvals, hipMemcpyHostToDevice);

    hipblasStatus_t stat = hipblasDgemv(h, HIPBLAS_OP_N, nvals, nvals, alpha_, A_, nvals, x_, 1, beta_, y_, 1);

    if (stat == HIPBLAS_STATUS_SUCCESS)
    {
        printf("La chiamata a hipblasDgemv è stata eseguita con successo.\n");
    }
    else if (stat == HIPBLAS_STATUS_NOT_INITIALIZED)
    {
        printf("Errore: cuBLAS non è stato inizializzato correttamente.\n");
    }
    else if (stat == HIPBLAS_STATUS_INVALID_VALUE)
    {
        printf("Errore: uno o più parametri di input sono invalidi.\n");
    }
    else if (stat == HIPBLAS_STATUS_ALLOC_FAILED)
    {
        printf("Errore: l'allocazione di memoria sulla GPU ha fallito.\n");
    }

    hipMemcpy(y, y_, sz, hipMemcpyDeviceToHost);

    for (int i = 0; i < nvals; i++)
        printf("%f\n", y[i]);

    hipblasDestroy(h);

    hipFree(x_);
    hipFree(y_);
    hipFree(A_);
    hipFree(alpha_);
    hipFree(beta_);

    return 0;
}