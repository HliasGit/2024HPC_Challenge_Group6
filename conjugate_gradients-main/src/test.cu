#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

int main (void)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));

    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= M; i++) {
            a[IDX2F(i,j,M)] = (float)((i-1) * N + j);
        }
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("Memory allcation failed");
        printf ("Error value = %s ", cudaStat);

        return EXIT_FAILURE;
    }

    printf("OK !\n");
    return 0;
}