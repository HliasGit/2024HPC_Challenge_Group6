//Example 2. Application Using C and cuBLAS: 0-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 2
#define N 2


int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA, *devPtrb, *devPtrc;
    float* a = 0;
    float* b = 0, *c;
    int size = N;
    a = (float *) malloc (M * N * sizeof (*a));
    b = (float *) malloc (N * sizeof (*a));
    c = (float *) malloc (N * sizeof (*a));
    for (size_t i= 0; i<M; i++)
        for (size_t j=0; j<N; j++)
            a[i*N + j] = 1;
    for (size_t i= 0; i<M; i++)
        b[i] = 1.0;
    for (size_t i= 0; i<M; i++)
        c[i] = 1.0;

    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc ((void**)&devPtrA, N*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc ((void**)&devPtrb, N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc ((void**)&devPtrc, N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    stat = hipblasSetMatrix (M, N, sizeof(float), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        printf("Error in A\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipMemcpy(devPtrb, b, size * sizeof(float), hipMemcpyHostToDevice);
    // stat = cublasSetVector(size, sizeof(float), b, 1, devPtrb, 1);
    // if (stat != CUBLAS_STATUS_SUCCESS) {
    //     printf ("data download failed");
    //     printf("Error in b\n%d", stat);
    //     cudaFree (devPtrA);
    //     cublasDestroy(handle);
    //     return EXIT_FAILURE;
    // }
    hipMemcpy(devPtrc, c, size * sizeof(float), hipMemcpyHostToDevice);
    // stat = cublasSetVector (size, sizeof(float), c, 1, devPtrc, 1);
    // if (stat != CUBLAS_STATUS_SUCCESS) {
    //     printf ("data download failed");
    //     printf("Error in c\n");
    //     cudaFree (devPtrA);
    //     cublasDestroy(handle);
    //     return EXIT_FAILURE;
    // }
    float alpha = 1, *d_alpha;
    float beta = 1, *d_beta;
    cudaStat = hipMalloc ((void**)&d_alpha, sizeof(*a));
    cudaStat = hipMalloc ((void**)&d_beta, sizeof(*a));

    hipMemcpy(d_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, &beta,  sizeof(float), hipMemcpyHostToDevice);

    stat = hipblasSgemv(handle, HIPBLAS_OP_N, size, size, d_alpha, devPtrA, size, devPtrb, 1, d_beta, devPtrc, 1);

    hipMemcpy(b, devPtrb, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, devPtrc, size * sizeof(float), hipMemcpyDeviceToHost);
    printf("Stampiamo b:\n");
    for (j = 0; j < N; j++) {
        printf ("%7.0f", b[i]);
    }
    printf("\n");
    printf("Stampiamo c:\n");
    for (j = 0; j < N; j++) {
        printf ("%7.0f", c[i]);
    }
    printf("\n");
    if (stat == HIPBLAS_STATUS_SUCCESS) {
        printf("La chiamata a hipblasSgemv è stata eseguita con successo.\n");
    } else if (stat == HIPBLAS_STATUS_NOT_INITIALIZED) {
        printf("Errore: cuBLAS non è stato inizializzato correttamente.\n");
    } else if (stat == HIPBLAS_STATUS_INVALID_VALUE) {
        printf("Errore: uno o più parametri di input sono invalidi.\n");
    } else if (stat == HIPBLAS_STATUS_ALLOC_FAILED) {
        printf("Errore: l'allocazione di memoria sulla GPU ha fallito.\n");
    }


    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasGetMatrix (2, 0, sizeof(*a), devPtrb, M, b, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipMemcpy(devPtrb, b, size * sizeof(float), hipMemcpyDeviceToHost);
    // stat = cublasGetVector (M, sizeof(*a), devPtrb, 1, b, 1);
    // if (stat != CUBLAS_STATUS_SUCCESS) {
    //     printf ("data upload failed");
    //     cudaFree (devPtrA);
    //     cublasDestroy(handle);
    //     return EXIT_FAILURE;
    // }

    stat = hipblasGetVector (M, sizeof(*a), devPtrc, 1, c, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipFree (devPtrb);
    hipFree (devPtrc);
    hipblasDestroy(handle);
    printf("\n");
    free(a);
    return EXIT_SUCCESS;
}